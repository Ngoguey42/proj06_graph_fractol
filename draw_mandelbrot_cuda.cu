
#include <hip/hip_runtime.h>
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   draw_mandelbrot_cuda.c                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: ngoguey <ngoguey@student.42.fr>            +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2014/12/11 15:58:47 by ngoguey           #+#    #+#             */
/*   Updated: 2014/12/11 17:31:48 by ngoguey          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

/* #include <fractol.h> */

typedef unsigned char t_byte;

typedef union   u_co
{
    struct
    {
        t_byte  b;
        t_byte  g;
        t_byte  r;
        t_byte  a;
    }           s;
    int         i;
}               t_co;

typedef struct  s_cooi
{
    int         x;
    int         y;
    int         z;
}               t_cooi;

typedef struct  s_coof
{
    float       x;
    float       y;
    float       z;
}               t_coof;

typedef struct  s_img
{
    void        *img;
    char        *dat;
    int         bpp;
    int         lsz;
    int         end;
    int         totby;
}               t_img;

typedef struct  s_juli
{
    t_coof      m_coo;
    t_coof      curc;
}               t_juli;

typedef struct  s_mand
{

    int         idk;

}               t_mand;

typedef struct  s_fra
{
    void        *serv;
    void        *win1;
    t_img       s;
    int         ev[11];
    t_coof      coo;
    float       zoom;
    t_coof      scdt;
    t_coof      pxin;
    t_cooi      m_cooscr;
    int         type;
    t_juli      j;
    t_mand      m;
    int         part;
}               t_fra;

# define ABS(x) ((x) < 0 ? -(x) : (x))

# define VCOTOI(R, G, B, A) ((t_co){{B, G, R, A}})

# define WINY (float)652.

# define WIN_Y (int)(WINY)
# define WIN_X (int)(WINY * (float)1.0)

/* #include <string.h> */
/* #include <stdlib.h> */

/* #define LOL (80 * fra.zoom)  */
/* #define LOL2 (0.01 / fra.zoom) */
#define LOL (80 * 2.) 
#define LOL2 (0.01)

__device__ int fra_puts_dst_cuda(t_fra fra, int dst, t_co c, char *d_dat)
{
    int             i;

    i = -1;
    while (++i < (fra.s.bpp / 8))
    {
        d_dat[dst + i] = (char)c.s.b;
        c.i >>= 8 ;
    }
    return (0);

}


/* int		julia_zero_escape_val(t_fra fra, t_coof pix) */
__device__ int		julia_zero_escape_val(t_coof d_pix)
{
	t_coof	zero;
	t_coof	tmp;
	int		i;

/* 	ft_bzero(&zero, sizeof(t_coof)); */
	zero.x = 0;
	zero.y = 0;
	i = 0;
	while (i < LOL)
	{
		i++;
		tmp = zero;
		zero.y = 2 * tmp.x * tmp.y + d_pix.y;
		zero.x = tmp.x * tmp.x - tmp.y * tmp.y + d_pix.x;
		if (zero.x > 100. || zero.y > 100.)
			break;

		if (ABS(tmp.x - zero.x) < LOL2 && ABS(tmp.y - zero.y) < LOL2)
			return (LOL);
	}
	return (i);
}

/* void	fra_draw_mandelbrot2(t_fra fra) */
__global__ void	fra_draw_mandelbrot2(t_fra d_fra, char *d_dat)
{
	t_coof		pix;
	int			i;
	int			j;
	int			c;

	c = threadIdx.x;
	pix = d_fra.coo;
	i = -1 + (c - 1) * WIN_X;
	while (i < (c * WIN_X - 1))
	{
		i++;
		if (i % WIN_X == 0)
		{
			pix = d_fra.coo;
			pix.y += (float)(i / WIN_X) * d_fra.pxin.y;
		}
		else
			pix.x += d_fra.pxin.x;
		j = julia_zero_escape_val(pix);
		if (j < LOL)
			fra_puts_dst_cuda(d_fra, i * 4, VCOTOI(255, 0, 0, 0), d_dat);
		else
			fra_puts_dst_cuda(d_fra, i * 4, VCOTOI(0, 255, 0, 0), d_dat);
		
	}
}

int		fra_draw_mandelbrot(t_fra fra)
{
	t_fra		*d_fra;
/* 	char		d_dat[WIN_X * WIN_Y * 4]; */
	char		*d_dat;

	hipMalloc((void**)&d_fra, sizeof(t_fra));
	hipMalloc((void**)&d_dat, WIN_X * WIN_Y * sizeof(int));
	
	hipMemcpy(d_fra, &fra, sizeof(t_fra), hipMemcpyHostToDevice);

	fra_draw_mandelbrot2<<< 1, WIN_Y >>>(*d_fra, d_dat);


	return (0);
}
